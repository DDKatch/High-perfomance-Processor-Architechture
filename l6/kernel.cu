#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime_api.h"
#include <stdlib.h>
#include "utils.h"
#include <chrono>

typedef std::chrono::high_resolution_clock Clock;

#define ROW_SIZE 1024 // MATRIX = ROW_SIZE * ROW_SIZE
#define ROW_THREADS 32 // block threads = ROW_THREADS * ROW_THREADS

using namespace std;

void showMatrix(int* matrix){
	for (int i = 0; i < ROW_SIZE; i++){
		for (int j = 0; j < ROW_SIZE; j++){
			printf("%4.0d ", (matrix + i * ROW_SIZE)[j]);
		}
		printf("\n");
	}
}

int* initMatrix(){
	int* matrix = (int*)malloc(ROW_SIZE * ROW_SIZE * sizeof(int));
	for (int i = 0; i < ROW_SIZE * ROW_SIZE; i++)
		matrix[i] = 2;

	return matrix;
}

__global__ void matrixMulKernel(int* matr1, int* matr2, int* matr3, int* matr4){

	int y = blockDim.y*blockIdx.y + threadIdx.y;
	int x = blockDim.x*blockIdx.x + threadIdx.x;

	if (x < ROW_SIZE && y < ROW_SIZE){
		int offset = y*ROW_SIZE + x;
		int roffset = x*ROW_SIZE + y;
		matr1[offset] = matr1[offset] * matr2[roffset] + matr3[offset] * matr4[roffset];
	}
}

int* matrixMulAndSumCuda(int* matrix){

	int* d_matr1 = nullptr;
	int* d_matr2 = nullptr;
	int* d_matr3 = nullptr;
	int* d_matr4 = nullptr;

	dim3 blockSize;
	dim3 gridSize;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	matrix = initMatrix();
	int tmatr1[ROW_SIZE * ROW_SIZE];
	int tmatr2[ROW_SIZE * ROW_SIZE];
	int tmatr3[ROW_SIZE * ROW_SIZE];
	int tmatr4[ROW_SIZE * ROW_SIZE];

	for (int i = 0; i < ROW_SIZE; i++)
		for (int j = 0; j < ROW_SIZE; j++){
			tmatr1[i * ROW_SIZE + j] = matrix[i * ROW_SIZE + j];
			tmatr2[i * ROW_SIZE + j] = matrix[i * ROW_SIZE + j];
			tmatr3[i * ROW_SIZE + j] = matrix[i * ROW_SIZE + j];
			tmatr4[i * ROW_SIZE + j] = matrix[i * ROW_SIZE + j];
		}

	checkCudaErrors(hipSetDevice(0));
	//size_t pitch = ROW_SIZE*sizeof(int);          // size of row in bytes
	//checkCudaErrors(hipMallocPitch((void **)&dev_matrix, &pitch, (size_t)ROW_SIZE*sizeof(int), (size_t)ROW_SIZE));
	checkCudaErrors(hipMalloc((void**)&d_matr1, ROW_SIZE * ROW_SIZE * sizeof(int)));
	checkCudaErrors(hipMemcpy(d_matr1, tmatr1, ROW_SIZE * ROW_SIZE * sizeof(int), hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&d_matr2, ROW_SIZE * ROW_SIZE * sizeof(int)));
	checkCudaErrors(hipMemcpy(d_matr2, tmatr2, ROW_SIZE * ROW_SIZE * sizeof(int), hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&d_matr3, ROW_SIZE * ROW_SIZE * sizeof(int)));
	checkCudaErrors(hipMemcpy(d_matr3, tmatr3, ROW_SIZE * ROW_SIZE * sizeof(int), hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&d_matr4, ROW_SIZE * ROW_SIZE * sizeof(int)));
	checkCudaErrors(hipMemcpy(d_matr4, tmatr4, ROW_SIZE * ROW_SIZE * sizeof(int), hipMemcpyHostToDevice));
	
	blockSize = dim3(ROW_THREADS, ROW_THREADS, 1);
	gridSize = dim3(ROW_SIZE * ROW_SIZE / blockSize.x, ROW_SIZE * ROW_SIZE / blockSize.y, 1);

	hipEventRecord(start);
	matrixMulKernel <<< gridSize, blockSize >>> (d_matr1, d_matr2, d_matr3, d_matr4);
	hipEventRecord(stop);

	checkCudaErrors(hipMemcpy(matrix, d_matr1, ROW_SIZE * ROW_SIZE * sizeof(int), hipMemcpyDeviceToHost));
	hipEventSynchronize(stop);
	hipDeviceSynchronize();
	checkCudaErrors(hipGetLastError());

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	cout << "CUDA time simple (ms): " << milliseconds << endl;

	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipFree(d_matr1);
	hipFree(d_matr2);
	hipFree(d_matr3);
	hipFree(d_matr4);
	return matrix;
}

int* cpuMulAndSumMatr(int* matrix){
	
	int tmatr1[ROW_SIZE][ROW_SIZE];
	int tmatr2[ROW_SIZE][ROW_SIZE];
	int tmatr3[ROW_SIZE][ROW_SIZE];
	int tmatr4[ROW_SIZE][ROW_SIZE];

	for (int y = 0; y < ROW_SIZE; y++)
		for (int x = 0; x < ROW_SIZE; x++){
			tmatr1[y][x] = matrix[y*ROW_SIZE + x];
			tmatr2[y][x] = matrix[y*ROW_SIZE + x];
			tmatr3[y][x] = matrix[y*ROW_SIZE + x];
			tmatr4[y][x] = matrix[y*ROW_SIZE + x];
		}

	for (int y = 0; y < ROW_SIZE; y++)
		for (int x = 0; x < ROW_SIZE; x++){
			tmatr1[y][x] *= tmatr2[x][y];
			tmatr3[y][x] *= tmatr4[x][y];
			tmatr1[y][x] += tmatr3[y][x];
		}

	for (int y = 0; y < ROW_SIZE; y++)
		for (int x = 0; x < ROW_SIZE; x++)
			matrix[y*ROW_SIZE + x] = tmatr1[y][x];

	return matrix;
}

__host__ int main()
{
	int* matrix = initMatrix();
	int* cpumatrix = initMatrix();

	matrix = matrixMulAndSumCuda(matrix);
	checkCudaErrors(hipDeviceReset());

	//showMatrix(matrix);

	/*__int64 start;
	start = __rdtsc();

	cpumatrix = cpuMultMatr(cpumatrix);

	cout << "CPU time: " << __rdtsc() - start << '\n';*/

	auto start = Clock::now();

	cpumatrix = cpuMulAndSumMatr(cpumatrix);
	auto end = Clock::now();

	cout << "CPU time (ms): " << chrono::duration_cast<std::chrono::milliseconds>(end - start).count() << '\n';

	//showMatrix(cpumatrix);

	bool equal = true;
	for (int i = 0; i < ROW_SIZE; i++)
		for (int j = 0; j < ROW_SIZE; j++)
			if (matrix[i * ROW_SIZE + j] != cpumatrix[i * ROW_SIZE + j])
				equal = false;

	if (equal)
		cout << "equal\n";
	else
		cout << "not equal\n";

	system("pause");

    return 0;
}


//
//#include <iostream>
//#include <algorithm>
//using namespace std;
//#define N 1024
//#define RADIUS 3
//#define BLOCK_ROW_SIZE 16
//__global__ void stencil_1d(int *in, int *out) {
//	__shared__ int temp[BLOCK_ROW_SIZE + 2 * RADIUS];
//	int gindex = threadIdx.x + blockIdx.x * blockDim.x;
//	int lindex = threadIdx.x + RADIUS;
//	// Read input elements into shared memory
//	temp[lindex] = in[gindex];
//	if (threadIdx.x < RADIUS) {
//		temp[lindex - RADIUS] = in[gindex - RADIUS];
//		temp[lindex + BLOCK_ROW_SIZE] = in[gindex + BLOCK_ROW_SIZE];
//	}
//	// Synchronize (ensure all the data is available)
//	__syncthreads();
//	// Apply the stencil
//	int result = 0;
//	for (int offset = -RADIUS; offset <= RADIUS; offset++)
//		result += temp[lindex + offset];
//	// Store the result
//	out[gindex] = result;
//}
//void fill_ints(int *x, int n) {
//	fill_n(x, n, 1);
//}
//int main(void) {
//	int *in, *out; // host copies of a, b, c
//	int *d_in, *d_out; // device copies of a, b, c
//	int size = (N + 2 * RADIUS) * sizeof(int);
//	// Alloc space for host copies and setup values
//	in = (int *)malloc(size); fill_ints(in, N + 2 * RADIUS);
//	out = (int *)malloc(size); fill_ints(out, N + 2 * RADIUS);
//	// Alloc space for device copies
//	checkCudaErrors(hipMalloc((void **)&d_in, size));
//	checkCudaErrors(hipMalloc((void **)&d_out, size));
//	// Copy to device
//	checkCudaErrors(hipMemcpy(d_in, in, size, hipMemcpyHostToDevice));
//	checkCudaErrors(hipMemcpy(d_out, out, size, hipMemcpyHostToDevice));
//	// Launch stencil_1d() kernel on GPU
//	stencil_1d << <N / BLOCK_ROW_SIZE, BLOCK_ROW_SIZE >> >(d_in + RADIUS, d_out + RADIUS);
//	// Copy result back to host
//	checkCudaErrors(hipMemcpy(out, d_out, size, hipMemcpyDeviceToHost));
//	// Cleanup
//	free(in); free(out);
//	hipFree(d_in); hipFree(d_out);
//	return 0;
//}
